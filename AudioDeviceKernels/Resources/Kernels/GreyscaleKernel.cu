
#include <hip/hip_runtime.h>
extern "C" __global__ void GrayscaleKernel(unsigned char* data, int length, int intensity) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelIndex = idx * 3;  // RGB-Pixel haben 3 Byte

    if (pixelIndex + 2 < length) {
        // RGB-Werte auslesen
        unsigned char r = data[pixelIndex];
        unsigned char g = data[pixelIndex + 1];
        unsigned char b = data[pixelIndex + 2];

        // Grauwert berechnen (Helligkeit mit Gewichtung)
        unsigned char gray = (r * 0.299f + g * 0.587f + b * 0.114f) * (intensity / 255.0f);

        // RGB durch Grauwert ersetzen
        data[pixelIndex] = gray;
        data[pixelIndex + 1] = gray;
        data[pixelIndex + 2] = gray;
    }
}
